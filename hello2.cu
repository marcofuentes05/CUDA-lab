#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   : 
 To build use  : nvcc hello2.cu -o hello2 -arch=sm_20 
 ============================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello ()
{
  int myID = ( blockIdx.y * gridDim.x + 
               blockIdx.x ) * blockDim.x * blockDim.y * blockDim.z + 
               threadIdx.y * blockDim.x + 
               threadIdx.x; 
   if (myID>=131071) {
	 printf("z id %i", blockIdx.z);
      printf ("Hello world from %i\n", myID);
   }
}

int main ()
{
  dim3 g (8, 32);
  dim3 b (32, 16);
  hello <<< g, b >>> ();
  hipDeviceSynchronize ();
  return 0;
}
