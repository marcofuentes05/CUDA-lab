#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   : 
 To build use  : nvcc hello.cu -o hello -arch=sm_20
 ============================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello()
{
   int threadId = threadIdx.x + blockDim.x*threadIdx.y;
   if (threadId == 1023) {
   printf("Hello world\tTHREAD ID: %d\n", threadId);
   }
}

int main()
{
  hello<<<1,2048>>>();
  hipDeviceSynchronize();
  return 0;
}
